#include "hip/hip_runtime.h"
#include <io/error.h>
#include <sim/sim_struct.h>
#include <sim/kernel.h>

#include <stdio.h>
#include <stdbool.h>

static inline __device__ FluidVoxel_t* index_voxel(SimState_t* sim, int x, int y)
{
    return &(sim->voxels[y * (sim->params.dims.x) + x]);
}

__device__ void _setVoxel(FluidVoxel_t* v, float xVel, float yVel, float density){
    // Calculate intermediate values for lattice vectors
    float ux_times_3 = 3.0  * xVel;
    float uy_times_3 = 3.0  * yVel;
    float ux_squared = xVel * xVel;
    float uy_squared = yVel * yVel;
    float ux_times_uy_times_2 = xVel * yVel * 2.0;
    float u_squared = ux_squared + uy_squared;
    float u_squared_times_150pct = u_squared * 1.5;

    float zero_mag = 4.0/9.0;
    float nesw_mag = 1.0/9.0;
    float crnr_mag = 1.0/36.0;

    // Calculate lattice vectors
    v->lattice_vectors.zero = zero_mag*(1 - u_squared_times_150pct) * density;

    v->lattice_vectors.east  = nesw_mag*(1 + ux_times_3 + 4.5*ux_squared - u_squared_times_150pct) * density;
    v->lattice_vectors.west  = nesw_mag*(1 - ux_times_3 + 4.5*ux_squared - u_squared_times_150pct) * density;
    v->lattice_vectors.north = nesw_mag*(1 + uy_times_3 + 4.5*uy_squared - u_squared_times_150pct) * density;
    v->lattice_vectors.south = nesw_mag*(1 - uy_times_3 + 4.5*uy_squared - u_squared_times_150pct) * density;

    v->lattice_vectors.northeast = crnr_mag*(1 + ux_times_3 + uy_times_3 + 4.5*(u_squared + ux_times_uy_times_2) - u_squared_times_150pct) * density;
    v->lattice_vectors.southeast = crnr_mag*(1 + ux_times_3 - uy_times_3 + 4.5*(u_squared - ux_times_uy_times_2) - u_squared_times_150pct) * density;
    v->lattice_vectors.northwest = crnr_mag*(1 - ux_times_3 + uy_times_3 + 4.5*(u_squared - ux_times_uy_times_2) - u_squared_times_150pct) * density;
    v->lattice_vectors.southwest = crnr_mag*(1 - ux_times_3 - uy_times_3 + 4.5*(u_squared + ux_times_uy_times_2) - u_squared_times_150pct) * density;

    // Copy over other values
    v->density = density;
    v->velocity.x = xVel;
    v->velocity.y = yVel;
}

__global__ void InitializerKernel(SimState_t* state){
    // Initialize simulation state in parallel
    int myX = blockIdx.x * blockDim.x + threadIdx.x;
    int myY = blockIdx.y * blockDim.y + threadIdx.y;

    float imSize = state->params.dims.x * state->params.dims.y;

    if(myX < state->params.dims.x && myY < state->params.dims.y){
        FluidVoxel_t* v = index_voxel(state, myX, myY);
        _setVoxel(
            v, 
            (float)myX,
            (float)myY*-1.0f,
            1.0
        );
        // _setVoxel(
        //     v, 
        //     state->params.boundary_velocity.x,
        //     state->params.boundary_velocity.y,
        //     1.0
        // );
        v->curl = 0.0;
    }
}

__global__ void NaiveKernel(KERNEL_PARAMS){
    // Right now, the kernel just displays a test pattern
    int myX = blockIdx.x * blockDim.x + threadIdx.x;
    int myY = blockIdx.y * blockDim.y + threadIdx.y;

    if(myX < state->params.dims.x && myY < state->params.dims.y){
        FluidVoxel_t* v = index_voxel(state, myX, myY);
        if(myY*state->params.dims.x + myX <= 5*state->frame){
            _setVoxel(
                v, 
                (float)myY,
                (float)myX,
                1.0
            );
        }
    }
    int frame = state->frame;
    __syncthreads();
    state->frame = frame + 1;
}

