#include "hip/hip_runtime.h"
#include <io/error.h>
#include <sim/sim_struct.h>
#include <sim/kernel.h>

#include <sim/_kernel_util.h>

#include <stdio.h>
#include <stdbool.h>

#undef TILE_WIDTH
#define TILE_WIDTH 32
#define TILE_HEIGHT 8
#define TILE_OVERLAP 1

#define IM_X ((signed int)blockIdx.x * ((signed int)blockDim.x-2*TILE_OVERLAP) + (signed int)threadIdx.x - TILE_OVERLAP)
#define IM_Y ((signed int)blockIdx.y * ((signed int)blockDim.y-2*TILE_OVERLAP) + (signed int)threadIdx.y - TILE_OVERLAP)
#define IM_BASE_X ((signed int)blockIdx.x * ((signed int)blockDim.x-2*TILE_OVERLAP) - TILE_OVERLAP)
#define IM_OFFS(sim)(_VOXEL_IDX(sim, IM_X, IM_Y))

#define MY_TILE_OFFS (threadIdx.y * blockDim.x + threadIdx.x)

static inline __device__ FluidVoxel_t* _index_tile_voxel(FluidVoxel_t* tile, int xOffs, int yOffs){
    int offs = (threadIdx.y + yOffs)*blockDim.x + threadIdx.x + xOffs;
    return tile + offs;
}

static __device__ void _collide(SimState_t* state, int myX, int myY, FluidVoxel_t* oldVA){
    if(myX < state->params.dims.x && myY < state->params.dims.y && myX >= 0 && myY >= 0){
        const float omega = 1.0f / ((3.0f*state->params.viscosity) + 0.5f);

        FluidVoxel_t* myVoxel = _index_tile_voxel(oldVA, 0, 0);

        // if(!isShared(myVoxel)){
        //     printf("Tile coordinates (%d,%d), Pointer: %x\n")
        // }

        // printf("Pointer: %x\n", myVoxel);

        // float f = myVoxel->density;

        const static float ux_dirs[] = {0,-1,-1,-1, 0, 1, 1, 1, 0};
        const static float uy_dirs[] = {1, 1, 0,-1,-1,-1, 0, 1, 0};
        float rho = 0.0f, ux = 0.0f, uy = 0.0f;
        for(int i = 0; i < NUM_LATTICE_VECTORS; ++i){
            float lv = myVoxel->lattice_vectors.sequence[i];
            rho += lv;
            ux  += lv * ux_dirs[i];
            uy  += lv * uy_dirs[i];
        }

        ux /= rho;
        uy /= rho;

        myVoxel->density = rho;
        myVoxel->velocity.x = ux;
        myVoxel->velocity.y = uy;

        _mod_voxel_lattice_vectors(myVoxel, ux, uy, rho, omega);
    }
}

static __device__ void _stream(SimState_t* state, int myX, int myY, FluidVoxel_t* oldVA, FluidVoxel_t* newVA){
    if(
        myX < state->params.dims.x && myY < state->params.dims.y && myY >= 0 && myX >= 0 // Check image dimensions
        && threadIdx.x > 0 && threadIdx.x < blockDim.x - 1 // Exclude edge voxels (x)
        && threadIdx.y > 0 && threadIdx.y < blockDim.y - 1 // Exclude edge voxels (y)
    ){
        FluidVoxel_t* myVoxel = _index_tile_voxel(newVA, 0, 0);
        FluidVoxel_t* myOldVoxel = _index_tile_voxel(oldVA, 0, 0);

        myVoxel->density    = myOldVoxel->density;
        myVoxel->velocity.x = myOldVoxel->velocity.x;
        myVoxel->velocity.y = myOldVoxel->velocity.y;

        // ========= Copy lattice vectors =========
        // Copy from voxel to the south
        myVoxel->lattice_vectors.named.north 
            = _index_tile_voxel(oldVA, 0, -1)
                -> lattice_vectors.named.north;

        // --> South
        // Copy from the voxel to the north
        myVoxel->lattice_vectors.named.south
            = _index_tile_voxel(oldVA, 0, 1)
                -> lattice_vectors.named.south;

        // --> East
        // Copy from the voxel to the west
        myVoxel->lattice_vectors.named.east
            = _index_tile_voxel(oldVA, -1, 0)
                -> lattice_vectors.named.east;

        // --> West
        // Copy from the voxel to the east
        myVoxel->lattice_vectors.named.west
            = _index_tile_voxel(oldVA, 1, 0)
                -> lattice_vectors.named.west;

        // --> Northeast
        // Copy from the voxel to the southwest
        myVoxel->lattice_vectors.named.northeast
            = _index_tile_voxel(oldVA, -1, -1)
                -> lattice_vectors.named.northeast;

        // --> Northwest
        // Copy from the voxel to the southeast
        myVoxel->lattice_vectors.named.northwest
            = _index_tile_voxel(oldVA, 1, -1)
                -> lattice_vectors.named.northwest;

        // --> Southeast
        // Copy from the voxel to the northwest
        myVoxel->lattice_vectors.named.southeast
            = _index_tile_voxel(oldVA, -1, 1)
                -> lattice_vectors.named.southeast;

        // --> Southwest
        // Copy from the voxel to the northeast
        myVoxel->lattice_vectors.named.southwest
            = _index_tile_voxel(oldVA, 1, 1)
                -> lattice_vectors.named.southwest;
    }
}

static __device__ void _barrierBounceBack(SimState_t* state, int myX, int myY, FluidVoxel_t* newVA){
    if(        
        myX < state->params.dims.x && myY < state->params.dims.y && myY >= 0 && myX >= 0 // Check image dimensions
        && threadIdx.x > 0 && threadIdx.x < blockDim.x - 1 // Exclude edge voxels (x)
        && threadIdx.y > 0 && threadIdx.y < blockDim.y - 1 // Exclude edge voxels (y)
    ){
        FluidVoxel_t* myVoxel = _index_tile_voxel(newVA, 0, 0);
        if(myVoxel->is_barrier){
            // For every lattice vector except the zero vector
            for(int i = 0; i < LV_IM; ++i){
                // Find the voxel in the opposite direction (i.e. if i == LV_S,
                // get the voxel to the north)
                IntPoint_t voxelDelta = _voxel_delta_in_direction(LV_OPPOSITE_DIR_OF(i));

                // Bounce back
                _index_tile_voxel(newVA, voxelDelta.x, voxelDelta.y)
                    -> lattice_vectors.sequence[LV_OPPOSITE_DIR_OF(i)]
                        = myVoxel->lattice_vectors.sequence[i];
            }
        }
    }

    if(myX == 0 && myY == 0){
        ++(state->frame);
    }
}

static __device__ void _parl_copy(void* dst, void* src, size_t len, size_t numThreads, size_t myTid){
    // int passCount = (len + numThreads - 1) / numThreads;
    int passCount = len / numThreads;

    for(int i = 0; i < passCount; ++i){
        int myOffs = i*numThreads + myTid;
        char x = ((char*)src)[myOffs];
        ((char*)dst)[myOffs] = x;
    }

    int myOffs = passCount*numThreads + myTid;
    if(myOffs < len){
        char x = ((char*)src)[myOffs];
        ((char*)dst)[myOffs] = x;
    }
}

static __device__ void _populateRow(SimState_t* state, FluidVoxel_t* rowPtr){

    if(IM_Y >= 0 && IM_Y < state->params.dims.y){
        FluidVoxel_t* dstPtr = rowPtr;
        FluidVoxel_t* srcPtr = state->_d_voxels_old + IM_Y*state->params.dims.x + IM_BASE_X;
        int copyLen = blockDim.x;

        if(blockIdx.x == 0){
            // We're on the left side, so we have to handle edge conditions
            dstPtr += 1;
            srcPtr += 1;
            copyLen -= 1;
        }else if(blockIdx.x == gridDim.x - 1){
            // We're on the right side, so we have to handle edge conditions
            copyLen -= 1;
        }

        // Perform copy operation
        // memcpy(dstPtr, srcPtr, copyLen*sizeof(FluidVoxel_t));
        _parl_copy(dstPtr, srcPtr, copyLen*sizeof(FluidVoxel_t), blockDim.x, threadIdx.x);
    }

    // Handle edge conditions while we wait for the row to copy
    FluidVoxel_t edgeCondition;

    _setVoxel(
        &edgeCondition, 
        state->params.boundary_velocity.x,
        state->params.boundary_velocity.y,
        1.0f
    );

    if(IM_X >= state->params.dims.x || IM_X < 0 || IM_Y >= state->params.dims.y || IM_Y < 0){
        rowPtr[threadIdx.x] = edgeCondition;
    }

    __syncwarp();
}

static __device__ void _writeBackRow(SimState_t* state, FluidVoxel_t* rowPtr){
    if(IM_Y >= 0 && IM_Y < state->params.dims.y){
        FluidVoxel_t* srcPtr = rowPtr;
        FluidVoxel_t* dstPtr = state->voxels + IM_Y*state->params.dims.x + IM_BASE_X;
        int copyLen = blockDim.x;

        if(blockIdx.x == 0){
            // We're on the left side, so we have to handle edge conditions
            dstPtr += 1;
            srcPtr += 1;
            copyLen -= 1;
        }else if(blockIdx.x == gridDim.x - 1){
            // We're on the right side, so we have to handle edge conditions
            copyLen -= 1;
        }

        // Perform copy operation
        if(threadIdx.x > 0 && threadIdx.x < blockDim.x - 1 && threadIdx.y > 0 && threadIdx.y < blockDim.y - 1){
            // memcpy(dstPtr, srcPtr, copyLen*sizeof(FluidVoxel_t));
            _parl_copy(dstPtr, srcPtr, copyLen*sizeof(FluidVoxel_t), blockDim.x - 2, threadIdx.x - 1);
        }
        __syncwarp();
    }
}

static __device__ void _tileCpy_g2s(SimState_t* state, FluidVoxel_t* tile){
    FluidVoxel_t edgeCondition;

    _setVoxel(
        &edgeCondition, 
        state->params.boundary_velocity.x,
        state->params.boundary_velocity.y,
        1.0f
    );

    if(IM_X >= 0 && IM_X < state->params.dims.x && IM_Y >= 0 && IM_Y < state->params.dims.y){
        tile[MY_TILE_OFFS] = state->voxels[IM_OFFS(state)];
    }else{
        tile[MY_TILE_OFFS] = edgeCondition;
    }
    __syncwarp();
}

static __device__ void _tileCpy_s2g(SimState_t* state, FluidVoxel_t* tile){
    FluidVoxel_t edgeCondition;

    _setVoxel(
        &edgeCondition, 
        state->params.boundary_velocity.x,
        state->params.boundary_velocity.y,
        1.0f
    );

    if(IM_X >= 0 && IM_X < state->params.dims.x && IM_Y >= 0 && IM_Y < state->params.dims.y){
        if(!(threadIdx.x == 0 || threadIdx.x == blockDim.x - 1 || threadIdx.y == 0 || threadIdx.y == blockDim.y - 1)){
            state->voxels[IM_OFFS(state)] = tile[MY_TILE_OFFS];
        }
    }
    __syncwarp();
}

static __device__ void _run(SimState_t* state){
    __shared__ FluidVoxel_t oldVA[TILE_WIDTH*TILE_HEIGHT];
    __shared__ FluidVoxel_t newVA[TILE_WIDTH*TILE_HEIGHT];

    // if(threadIdx.y == 0){
    //     // oldVA[0].lattice_vectors.named.north = 1.0f;
    //     memcpy(&oldVA, state->voxels, 5*sizeof(FluidVoxel_t));
    //     // _parl_copy(oldVA, state->voxels, 5*sizeof(FluidVoxel_t)/sizeof(int32_t), 32, threadIdx.x);
    // }

    __syncthreads();

    FluidVoxel_t* tileRowPtr = &(oldVA[threadIdx.y*TILE_WIDTH]);
    FluidVoxel_t* tileRowPtr_new = &(newVA[threadIdx.y*TILE_WIDTH]);

    // _populateRow(state, tileRowPtr);              // 1: Load data from global memory into oldVA
    _tileCpy_g2s(state, &(oldVA[0]));
    __syncthreads();
    // if(state->frame == 32){
    //     oldVA[MY_TILE_OFFS].velocity.x = (float)threadIdx.x;
    //     oldVA[MY_TILE_OFFS].velocity.y = (float)threadIdx.y;
    // }
    // __syncthreads();
    // No __syncthreads necessary here - see https://github.com/johnMamish/nuce468-w22-fluidsim/issues/3#issuecomment-1062043483
    _collide(state, IM_X, IM_Y, oldVA);           // 2: Collide step
    __syncthreads();                              // 3: Synchronize
    _stream(state, IM_X, IM_Y, oldVA, newVA);     // 4: Stream step
    __syncthreads();                              // 5: Synchronize
    // _barrierBounceBack(state, IM_X, IM_Y, newVA); // 6: Barrier Bounce-Back step
    // __syncthreads();                              // 7: Synchronize
    // // _writeBackRow(state, tileRowPtr_new);                  // 8: Write newVA back to global memory
    _tileCpy_s2g(state, &(newVA[0]));

    if(IM_X == 0 && IM_Y == 0){
        ++(state->frame);
    }
}

__global__ void Opt2Kernel_Full(KERNEL_PARAMS){
    // Run all steps at once
    _run(state);
}

